#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <Windows.h>

__global__ void hello()
{
	printf("Hello World! I am thread number %d\n", blockIdx.x);
}

int main()
{
	hello << <16, 1 >> > ();

	hipDeviceSynchronize();

	printf("That's all folks.\n");

	system("pause");
    return 0;
}
