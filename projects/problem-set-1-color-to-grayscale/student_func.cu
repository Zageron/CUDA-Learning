#include "hip/hip_runtime.h"
// Homework 1
// Color to Greyscale Conversion

//A common way to represent color images is known as RGBA - the color
//is specified by how much Red, Green and Blue is in it.
//The 'A' stands for Alpha and is used for transparency, it will be
//ignored in this homework.

//Each channel Red, Blue, Green and Alpha is represented by one byte.
//Since we are using one byte for each color there are 256 different
//possible values for each color.  This means we use 4 bytes per pixel.

//Greyscale images are represented by a single intensity value per pixel
//which is one byte in size.

//To convert an image from color to grayscale one simple method is to
//set the intensity to the average of the RGB channels.  But we will
//use a more sophisticated method that takes into account how the eye 
//perceives color and weights the channels unequally.

//The eye responds most strongly to green followed by red and then blue.
//The NTSC (National Television System Committee) recommends the following
//formula for color to greyscale conversion:

//I = .299f * R + .587f * G + .114f * B

//Notice the trailing f's on the numbers which indicate that they are 
//single precision floating point constants and not double precision
//constants.

//You should fill in the kernel as well as set the block and grid sizes
//so that the entire image is processed.

#include "utils.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include ""

typedef unsigned int uint;

__global__
void rgba_to_greyscale(const uchar4* const rgbaImage,
                       unsigned char* const greyImage,
                       int numRows, int numCols)
{
    uint x = (blockIdx.x * blockDim.x) + threadIdx.x;
    uint y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < numCols && y < numRows)
    {
        // Future Comprehension
        // Since this is a single dimensional array, we need to multiply the 2D length of the array, how many columns are there.
        // So if we are on row 1, we need to add the entire number of columns to the index and then add the actual column (in 1D) we are on.
        uint idx = (y * numCols) + x;
        greyImage[idx] = rgbaImage[idx].x * 0.299f + rgbaImage[idx].y * 0.587f + rgbaImage[idx].z * 0.114f;
    }
}

void your_rgba_to_greyscale(const uchar4 * const h_rgbaImage, uchar4 * const d_rgbaImage,
                            unsigned char* const d_greyImage, size_t numRows, size_t numCols)
{
    const dim3 threadsPerBlock(32, 32);
    const dim3 numBlocks((numCols + threadsPerBlock.y) / threadsPerBlock.y + 1, (numRows + threadsPerBlock.x) / threadsPerBlock.x);

    rgba_to_greyscale << <numBlocks, threadsPerBlock >> > (d_rgbaImage, d_greyImage, numRows, numCols);

    hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
}
