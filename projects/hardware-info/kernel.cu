#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <iostream>

#include "thrust\host_vector.h"
#include "thrust\device_vector.h"

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	int computeCompatabilityMajor;
	int computeCompatabilityMinor;
	hipDeviceGetAttribute(&computeCompatabilityMajor, hipDeviceAttributeComputeCapabilityMajor, 0);
	hipDeviceGetAttribute(&computeCompatabilityMinor, hipDeviceAttributeComputeCapabilityMinor, 0);
	printf("Compute Version: %d.%d\n", computeCompatabilityMajor, computeCompatabilityMinor);


	int maxGridDimX;
	int maxGridDimY;
	int maxGridDimZ;
	hipDeviceGetAttribute(&maxGridDimX, hipDeviceAttributeMaxGridDimX, 0);
	hipDeviceGetAttribute(&maxGridDimY, hipDeviceAttributeMaxGridDimY, 0);
	hipDeviceGetAttribute(&maxGridDimZ, hipDeviceAttributeMaxGridDimZ, 0);
	printf("Max Grid Dimensions: (%d, %d, %d)\n", maxGridDimX, maxGridDimY, maxGridDimZ);


	int maxBlockDimX;
	int maxBlockDimY;
	int maxBlockDimZ;
	hipDeviceGetAttribute(&maxBlockDimX, hipDeviceAttributeMaxBlockDimX, 0);
	hipDeviceGetAttribute(&maxBlockDimY, hipDeviceAttributeMaxBlockDimY, 0);
	hipDeviceGetAttribute(&maxBlockDimZ, hipDeviceAttributeMaxBlockDimZ, 0);
	printf("Max Block Dimensions: (%d, %d, %d)\n", maxBlockDimX, maxBlockDimY, maxBlockDimZ);


	int maxThreads;
	hipDeviceGetAttribute(&maxThreads, hipDeviceAttributeMaxThreadsPerBlock, 0);
	printf("Max Threads per Block: %d\n", maxThreads);


	system("pause");
}
